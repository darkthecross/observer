#include "hip/hip_runtime.h"
#include "gpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace gpu_util {

namespace {

#define IMG_W 848
#define IMG_H 480

__global__ void FilterPixel(unsigned char* img, unsigned char thres)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i >= IMG_W * IMG_H) return;
  if (img[i] < thres) {
    img[i] = 0;
  } else {
    img[i] = 255;
  }
}

// TODO(darkthecross): Use shared memory.
__global__ void MeanShiftIteration(unsigned char* img, unsigned char* out_img, int kernel_size, float diff_thresh) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int img_y = i % IMG_W;
  int img_x = i / IMG_W;
  int max_offset = kernel_size / 2;
  unsigned int num_valid_pixels = 0;
  int avg_pixel = 0;
  for(int xx = -max_offset; xx <= max_offset; ++xx) {
    for(int yy = -max_offset; yy <= max_offset; ++yy) {
      int xp = img_x + xx;
      int yp = img_y + yy;
      if(xp >= 0 && xp < IMG_H && yp >= 0 && yp < IMG_W) {
        unsigned char cur_pixel = *(img + xp * IMG_W + yp);
        if( cur_pixel > 0 ) {
          ++num_valid_pixels;
          avg_pixel += int(cur_pixel);
        }
      }
    }
  }
  avg_pixel = float(avg_pixel) / float(num_valid_pixels);
  if( (float)abs( int(*(img+i)) - avg_pixel ) / (float)*(img+i) < diff_thresh ) {
    *(out_img + i) = avg_pixel;
  } else {
    *(out_img + i) = *(img+i);
  }
}

__global__ void CopyDeviceMemory(unsigned char* from, unsigned char* to, int* diff_counter) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i >= IMG_W * IMG_H) return;
  if(*(to + i) != *(from + i)) {
    atomicAdd(diff_counter, 1);
  }
  *(to + i) = *(from + i);
}



}  // namespace

void ThresholdImage(unsigned char* in_mat, unsigned char* out_mat, unsigned char thresh) {
  unsigned char *device_mat;

  hipMalloc(&device_mat, IMG_W*IMG_H*sizeof(unsigned char)); 
  hipMemcpy(device_mat, in_mat, IMG_W*IMG_H*sizeof(unsigned char), hipMemcpyHostToDevice);

  FilterPixel<<<(IMG_W*IMG_H+255)/256, 256>>>(device_mat, thresh);

  hipMemcpy(out_mat, device_mat, IMG_W*IMG_H*sizeof(unsigned char), hipMemcpyDeviceToHost);
}

void MeanShiftSegmentation(unsigned char* in_mat, unsigned char* out_mat, int kernel_size, int num_max_iter, float diff_thresh) {
  unsigned char *device_mat, *iter_mat;

  hipMalloc(&device_mat, IMG_W*IMG_H*sizeof(unsigned char)); 
  hipMemcpy(device_mat, in_mat, IMG_W*IMG_H*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMalloc(&iter_mat, IMG_W*IMG_H*sizeof(unsigned char)); 

  int* device_diff_counter;
  hipMalloc(&device_diff_counter, sizeof(int)); 
  int host_diff;
  
  for(int i = 0; i<num_max_iter; ++i) {
    hipMemset(device_diff_counter, 0, sizeof(int));
    MeanShiftIteration<<<(IMG_W*IMG_H+255)/256, 256>>>(device_mat, iter_mat, kernel_size, diff_thresh);
    CopyDeviceMemory<<<(IMG_W*IMG_H+255)/256, 256>>>(iter_mat, device_mat, device_diff_counter);
    hipMemcpy(&host_diff, device_diff_counter, sizeof(int), hipMemcpyDeviceToHost);
  }

  hipMemcpy(out_mat, device_mat, IMG_W*IMG_H*sizeof(unsigned char), hipMemcpyDeviceToHost);
}


}  // namespace gpu_util